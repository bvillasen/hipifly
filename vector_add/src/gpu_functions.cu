
#include "stdio.h"

#ifndef ENABLE_HIP
#include <hip/hip_runtime.h>
#else
// Translate CUDA functions to the analogous HIP function
#include "cuda_to_hip.h"
#endif

#define TPB 256 //Number of threads per block for kernel calls

void get_device_properties( const int device_id ){
 	hipDeviceProp_t prop;
	hipGetDeviceProperties( &prop, device_id );
  printf("Device: %d  name:  %s \n", device_id, prop.name );

}


int set_device( int device_id ){
  
  int n_devices;
  hipGetDeviceCount(&n_devices);
  printf("Number of available devices %d\n", n_devices);  
  printf("Device id: %d \n", device_id);
  if ( device_id >= n_devices ){
    printf( "ERROR: Device %d is not available. Only %d devices detected.\n", device_id, n_devices );
    return -1;
  }
  
  hipSetDevice(device_id); 
  get_device_properties( device_id );

  return 0;

}

void allocate_device_arrays( int N, double *&d_a, double *&d_b, double *&d_c  ){
  hipMalloc( (void **)&d_a, N*sizeof(double) );
  hipMalloc( (void **)&d_b, N*sizeof(double) );
  hipMalloc( (void **)&d_c, N*sizeof(double) );  
}


void copy_host_to_device( int N, double *h_a, double *h_b, 
                          double *&d_a, double *&d_b   ){
  hipMemcpy( d_a, h_a, N*sizeof(double), hipMemcpyHostToDevice );
  hipMemcpy( d_b, h_b, N*sizeof(double), hipMemcpyHostToDevice );
}


__global__ void vector_add_kernel( int N, double *d_a, double *d_b, double *d_c ){
  int tid = threadIdx.x + blockIdx.x * blockDim.x;

  if ( tid < N ){
    d_c[tid] = d_a[tid] + d_b[tid];
  }
}

float gpu_vector_add( int N, double *d_a, double *d_b, double *d_c ){

  int n_grid = ( N - 1 )/TPB + 1;
  dim3 grid( n_grid, 1, 1 );
  dim3 block( TPB, 1, 1 );

  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);

  hipEventRecord(start);
  vector_add_kernel<<<grid, block, 0, 0>>>( N, d_a, d_b, d_c );
  hipEventRecord(stop);
  hipEventSynchronize(stop);
  
  float elapsed_time_milliseconds = 0;
  hipEventElapsedTime(&elapsed_time_milliseconds, start, stop);
  return elapsed_time_milliseconds;

}


void copy_device_to_host( int N, double *d_a, double *h_a ){
  hipMemcpy( h_a, d_a, N*sizeof(double), hipMemcpyDeviceToHost);
}